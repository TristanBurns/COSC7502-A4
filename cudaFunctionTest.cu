
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <stdlib.h>
#include <chrono>


#define p(I, J) *((p + (I)*n) + (J))

//######################################################################################
//####                                                                              ####
//####                               User Input (macros)                            ####
//####                                                                              ####
//######################################################################################

#define N 16                    // Chromosome size (number of genes per individual).
#define M 8                     // Population size (number of individuals).
//#define blockSize 8                     // Population size (number of individuals)
#define seed 42                 // Psuedorandom number generator seed (std:srand(seed)).
#define maxgenerations 2000     // Maximum number of generations (while loop limit).
#define t 2                     // Tournament size (parents competing for selection).
#define verbose true            // Verbose output (cout) for verification.
#define printfitness false      // Best fitness per generation output (cout).

//######################################################################################
//####                                                                              ####
//####                         DO NOT MODIFY BELOW THIS LINE                        ####
//####                                                                              ####
//######################################################################################

// global variables to store the matrix

int* P = nullptr;
int* F = nullptr;
int* pDevice;
int* fDevice;

void PrintPopulationVerbose(int *p, int n, int m)
/*
Fill static 2d int array at pointer *p with m indivduals with n
chromosmomes, gene values 0 or 1 (verbose cout).
*/
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            std::cout << p(i, j);
        }
        std::cout << " -> Individual " << i << std::endl;
    }
    return;
}



void RandomPopulationVerbose(int *p, int n, int m)
/*
Fill static 2d int array at pointer *p with m indivduals with n
chromosmomes, gene values 0 or 1 (verbose cout).
*/
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            p(i, j) = ((int)std::rand() % 2);
            std::cout << p(i, j);
        }
        std::cout << " -> Individual " << i << std::endl;
    }
    return;
}


int MaxFitnessVerbose(int *p, int n, int m)
/*
Return integer of best (highest) fitness for the pointer *p with m indivduals with n
chromosmomes. Fitness is defined as the sum of the individuals chromosomes (verbose cout).
*/
{
    int best = 0;
    int besti = 0;
    int sum = 0;
    std::cout << "Running MaxFitness (Verbose)" << std::endl;
    for (int i = 0; i < m; i++)
    {
        sum = 0;
        for (int j = 0; j < n; j++)
        {
            std::cout << p(i, j);
            sum += p(i, j);
        }
        std::cout << " -> Individual " << i << ", Fitness: " << sum << std::endl;

        if (sum > best)
        {
            best = sum;
            besti = i;
        }
    }
    return best;
}



void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}


template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, unsigned int n) {
extern __shared__ int sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockSize*2) + tid;
unsigned int gridSize = blockSize*2*gridDim.x;
sdata[tid] = 0;
while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
__syncthreads();
if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) warpReduce<blockSize>(sdata, tid);
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}






int main()
{
    std::srand(seed);
    P = static_cast<int*>(malloc(N*M*sizeof(int)));
    F = static_cast<int*>(malloc(N*M*sizeof(int)));
    RandomPopulationVerbose(P, N,  M);
    MaxFitnessVerbose(P, N,  M);

    checkError(hipMalloc(&pDevice, N*M*sizeof(int)));
    checkError(hipMalloc(&fDevice, N*M*sizeof(int)));
    checkError(hipMemcpy(pDevice, P, sizeof(double)*N*M, hipMemcpyHostToDevice));

    int ThreadsPerBlock = 256;
    int Blocks = (N+ThreadsPerBlock-1)/ThreadsPerBlock;
    reduce6<4><<<Blocks,ThreadsPerBlock>>>(pDevice, fDevice, N);
    checkError(hipMemcpy(F, fDevice, N*sizeof(double), hipMemcpyDeviceToHost));
    PrintPopulationVerbose(F, N,  M);
    free(P);
    free(F);
    checkError(hipFree(pDevice));
    checkError(hipFree(fDevice));
    return 0;
}

